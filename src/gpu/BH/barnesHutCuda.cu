#include "hip/hip_runtime.h"
/*
   Copyright 2023 Hsin-Hung Wu

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <iostream>
#include <cmath>
#include "barnesHut_kernel.cuh"
#include "constants.h"
#include "err.h"

BarnesHutCuda::BarnesHutCuda(int n) : nBodies(n)
{
    nNodes = MAX_NODES;
    leafLimit = MAX_NODES - N_LEAF;
    h_b = new Body[n];
    h_node = new Node[nNodes];

    CHECK_CUDA_ERROR(hipMalloc((void **)&d_b, sizeof(Body) * n));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_node, sizeof(Node) * nNodes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_mutex, sizeof(int) * nNodes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_b_buffer, sizeof(Body) * n));
}

BarnesHutCuda::~BarnesHutCuda()
{

    delete[] h_b;
    delete[] h_node;
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_node));
    CHECK_CUDA_ERROR(hipFree(d_mutex));
    CHECK_CUDA_ERROR(hipFree(d_b_buffer));
}

void BarnesHutCuda::resetCUDA()
{
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nNodes / blockSize);
    ResetKernel<<<gridSize, blockSize>>>(d_node, d_mutex, nNodes, nBodies);
}
void BarnesHutCuda::computeBoundingBoxCUDA()
{
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ComputeBoundingBoxKernel<<<gridSize, blockSize>>>(d_node, d_b, d_mutex, nBodies);
}
void BarnesHutCuda::constructQuadTreeCUDA()
{
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ConstructQuadTreeKernel<<<1, blockSize>>>(d_node, d_b, d_b_buffer, 0, nNodes, nBodies, leafLimit);
}

void BarnesHutCuda::computeForceCUDA()
{
    int blockSize = 32;
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ComputeForceKernel<<<gridSize, blockSize>>>(d_node, d_b, nNodes, nBodies, leafLimit);
}

void BarnesHutCuda::initRandomBodies()
{
    srand(time(NULL));
    double maxDistance = MAX_DIST;
    double minDistance = MIN_DIST;
    Vector centerPos = {CENTERX, CENTERY};
    for (int i = 0; i < nBodies - 1; ++i)
    {

        double angle = 2 * M_PI * (rand() / (double)RAND_MAX);
        // Generate random distance from center within the given max distance
        double radius = (maxDistance - minDistance) * (rand() / (double)RAND_MAX) + minDistance;

        // Calculate coordinates of the point
        double x = centerPos.x + radius * std::cos(angle);
        double y = centerPos.y + radius * std::sin(angle);
        Vector position = {x, y};
        h_b[i].isDynamic = true;
        h_b[i].mass = EARTH_MASS;
        h_b[i].radius = EARTH_DIA;
        h_b[i].position = position;
        h_b[i].velocity = {0.0, 0.0};
        h_b[i].acceleration = {0.0, 0.0};
    }
    h_b[nBodies - 1].isDynamic = false;
    h_b[nBodies - 1].mass = SUN_MASS;
    h_b[nBodies - 1].radius = SUN_DIA;
    h_b[nBodies - 1].position = centerPos;
    h_b[nBodies - 1].velocity = {0.0, 0.0};
    h_b[nBodies - 1].acceleration = {0.0, 0.0};
}

void BarnesHutCuda::initSpiralBodies()
{

    srand(time(NULL));
    double maxDistance = MAX_DIST;
    double minDistance = MIN_DIST;
    Vector centerPos = {CENTERX, CENTERY};
    for (int i = 0; i < nBodies - 1; ++i)
    {

        double angle = 2 * M_PI * (rand() / (double)RAND_MAX);
        // Generate random distance from center within the given max distance
        double radius = (maxDistance - minDistance) * (rand() / (double)RAND_MAX) + minDistance;

        // Calculate coordinates of the point
        double x = centerPos.x + radius * std::cos(angle);
        double y = centerPos.y + radius * std::sin(angle);
        Vector position = {x, y};

        double distance = sqrt(pow(x - centerPos.x, 2) + pow(y - centerPos.y, 2));
        Vector r = {position.x - centerPos.x, position.y - centerPos.y};
        Vector a = {r.x / distance, r.y / distance};

        // Calculate velocity vector components
        double esc = sqrt((GRAVITY * SUN_MASS) / (distance));
        Vector velocity = {-a.y * esc, a.x * esc};

        h_b[i].isDynamic = true;
        h_b[i].mass = EARTH_MASS;
        h_b[i].radius = EARTH_DIA;
        h_b[i].position = position;
        h_b[i].velocity = velocity;
        h_b[i].acceleration = {0.0, 0.0};
    }
    h_b[nBodies - 1].isDynamic = false;
    h_b[nBodies - 1].mass = SUN_MASS;
    h_b[nBodies - 1].radius = SUN_DIA;
    h_b[nBodies - 1].position = centerPos;
    h_b[nBodies - 1].velocity = {0.0, 0.0};
    h_b[nBodies - 1].acceleration = {0.0, 0.0};
}

void BarnesHutCuda::initCollideGalaxy()
{

    srand(time(NULL));
    double maxDistance = MAX_DIST / 4.0;
    double minDistance = MIN_DIST;
    Vector centerPos = {-NBODY_WIDTH / 6.0, CENTERY};

    int galaxy1 = nBodies / 2;

    for (int i = 0; i < galaxy1 - 1; ++i)
    {

        double angle = 2 * M_PI * (rand() / (double)RAND_MAX);
        // Generate random distance from center within the given max distance
        double radius = (maxDistance - minDistance) * (rand() / (double)RAND_MAX) + minDistance;

        // Calculate coordinates of the point
        double x = centerPos.x + radius * std::cos(angle);
        double y = centerPos.y + radius * std::sin(angle);
        Vector position = {x, y};

        double distance = sqrt(pow(x - centerPos.x, 2) + pow(y - centerPos.y, 2));
        Vector r = {position.x - centerPos.x, position.y - centerPos.y};
        Vector a = {r.x / distance, r.y / distance};

        // Calculate velocity vector components
        double esc = sqrt((GRAVITY * SUN_MASS) / (distance));
        Vector velocity = {-a.y * esc, a.x * esc};

        h_b[i].isDynamic = true;
        h_b[i].mass = EARTH_MASS;
        h_b[i].radius = EARTH_DIA;
        h_b[i].position = position;
        h_b[i].velocity = velocity;
        h_b[i].acceleration = {0.0, 0.0};
    }
    h_b[galaxy1 - 1].isDynamic = true;
    h_b[galaxy1 - 1].mass = SUN_MASS;
    h_b[galaxy1 - 1].radius = SUN_DIA;
    h_b[galaxy1 - 1].position = centerPos;
    h_b[galaxy1 - 1].velocity = {0.0, 0.0};
    h_b[galaxy1 - 1].acceleration = {0.0, 0.0};

    centerPos = {NBODY_WIDTH / 6.0, CENTERY};

    for (int i = galaxy1; i < nBodies - 1; ++i)
    {

        double angle = 2 * M_PI * (rand() / (double)RAND_MAX);
        // Generate random distance from center within the given max distance
        double radius = (maxDistance - minDistance) * (rand() / (double)RAND_MAX) + minDistance;

        // Calculate coordinates of the point
        double x = centerPos.x + radius * std::cos(angle);
        double y = centerPos.y + radius * std::sin(angle);
        Vector position = {x, y};

        double distance = sqrt(pow(x - centerPos.x, 2) + pow(y - centerPos.y, 2));
        Vector r = {position.x - centerPos.x, position.y - centerPos.y};
        Vector a = {r.x / distance, r.y / distance};

        // Calculate velocity vector components
        double esc = sqrt((GRAVITY * SUN_MASS) / (distance));
        Vector velocity = {-a.y * esc, a.x * esc};

        h_b[i].isDynamic = true;
        h_b[i].mass = EARTH_MASS;
        h_b[i].radius = EARTH_DIA;
        h_b[i].position = position;
        h_b[i].velocity = velocity;
        h_b[i].acceleration = {0.0, 0.0};
    }
    h_b[nBodies - 1].isDynamic = true;
    h_b[nBodies - 1].mass = SUN_MASS;
    h_b[nBodies - 1].radius = SUN_DIA;
    h_b[nBodies - 1].position = centerPos;
    h_b[nBodies - 1].velocity = {0.0, 0.0};
    h_b[nBodies - 1].acceleration = {0.0, 0.0};
}

void BarnesHutCuda::setBody(int i, bool isDynamic, double mass, double radius, Vector position, Vector velocity, Vector acceleration)
{
    h_b[i].isDynamic = isDynamic;
    h_b[i].mass = mass;
    h_b[i].radius = radius;
    h_b[i].position = position;
    h_b[i].velocity = velocity;
    h_b[i].acceleration = acceleration;
}

void BarnesHutCuda::initSolarSystem()
{
    setBody(0, true, 5.9740e24, 1.3927e6, {1.4960e11, 0}, {0, 2.9800e4}, {0, 0});
    setBody(1, true, 6.4190e23, 1.3927e6, {2.2790e11, 0}, {0, 2.4100e4}, {0, 0});
    setBody(2, true, 3.3020e23, 1.3927e6, {5.7900e10, 0}, {0, 4.7900e4}, {0, 0});
    setBody(3, true, 4.8690e24, 1.3927e6, {1.0820e11, 0}, {0, 3.5000e4}, {0, 0});
    setBody(4, false, 1.9890e30, 1.3927e6, {CENTERX, CENTERY}, {0, 0}, {0, 0});
}

Body *BarnesHutCuda::getBodies()
{

    return h_b;
}

void BarnesHutCuda::readDeviceBodies()
{
    CHECK_CUDA_ERROR(hipMemcpy(h_b, d_b, sizeof(Body) * nBodies, hipMemcpyDeviceToHost));
}

void BarnesHutCuda::setup(int sim)
{
    if (sim == 0)
    {
        initSpiralBodies();
    }
    else if (sim == 1)
    {
        initRandomBodies();
    }
    else if (sim == 2)
    {
        initCollideGalaxy();
    }
    else
    {
        initSolarSystem();
    }

    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, sizeof(Body) * nBodies, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_node, h_node, sizeof(Node) * nNodes, hipMemcpyHostToDevice));
}
void BarnesHutCuda::update()
{
    resetCUDA();
    computeBoundingBoxCUDA();
    constructQuadTreeCUDA();
    computeForceCUDA();
    CHECK_LAST_CUDA_ERROR();
}
