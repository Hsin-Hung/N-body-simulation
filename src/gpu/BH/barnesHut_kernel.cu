#include "hip/hip_runtime.h"
#ifndef BARNES_HUT_KERNEL_
#define BARNES_HUT_KERNEL_

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>
#include "constants.h"
#include "barnesHutCuda.cuh"
#include "barnesHut_kernel.cuh"

/*
----------------------------------------------------------------------------------------
RESET KERNEL
----------------------------------------------------------------------------------------
*/
__global__ void ResetKernel(Node *node, Vector *topLeft, Vector *botRight, int *mutex, int nNodes)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;

    if (b < nNodes)
    {

        node[b].centerMass = {-1, -1};
        node[b].totalMass = 0.0;
        node[b].isLeaf = true;
        mutex[b] = 0;
    }

    if (b == 0)
    {
        *topLeft = {INFINITY, -INFINITY};
        *botRight = {-INFINITY, INFINITY};
    }
}

/*
----------------------------------------------------------------------------------------
COMPUTE BOUNDING BOX
----------------------------------------------------------------------------------------
*/
__global__ void ComputeBoundingBox(Node *node, Body *bodies, Vector *topLeft, Vector *botRight, int *mutex, int nBodies)
{

    __shared__ double topLeftX[BLOCK_SIZE];
    __shared__ double topLeftY[BLOCK_SIZE];
    __shared__ double botRightX[BLOCK_SIZE];
    __shared__ double botRightY[BLOCK_SIZE];

    int tx = threadIdx.x;
    int b = blockIdx.x * blockDim.x + tx;

    topLeftX[tx] = INFINITY;
    topLeftY[tx] = -INFINITY;
    botRightX[tx] = -INFINITY;
    botRightY[tx] = INFINITY;

    __syncthreads();

    if (b < nBodies)
    {
        Body body = bodies[b];
        topLeftX[tx] = body.position.x;
        topLeftY[tx] = body.position.y;
        botRightX[tx] = body.position.x;
        botRightY[tx] = body.position.y;
    }

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tx < s)
        {
            topLeftX[tx] = fminf(topLeftX[tx], topLeftX[tx + s]);
            topLeftY[tx] = fmaxf(topLeftY[tx], topLeftY[tx + s]);
            botRightX[tx] = fmaxf(botRightX[tx], botRightX[tx + s]);
            botRightY[tx] = fminf(botRightY[tx], botRightY[tx + s]);
        }
    }

    if (tx == 0)
    {
        while (atomicCAS(mutex, 0, 1) != 0)
            ;
        topLeft->x = fminf(topLeft->x, topLeftX[0]);
        topLeft->y = fmaxf(topLeft->y, topLeftY[0]);
        botRight->x = fmaxf(botRight->x, botRightX[0]);
        botRight->y = fminf(botRight->y, botRightY[0]);
        atomicExch(mutex, 0);
    }
}

/*
----------------------------------------------------------------------------------------
CONSTRUCT QUAD TREE
----------------------------------------------------------------------------------------
*/

__device__ int getQuadrant(Vector topLeft, Vector botRight, double x, double y)
{

    if ((topLeft.x + botRight.x) / 2 >= x)
    {
        // Indicates topLeftTree
        if ((topLeft.y + botRight.y) / 2 <= y)
        {
            return 2;
        }
        // Indicates botLeftTree
        else
        {
            return 3;
        }
    }
    else
    {
        // Indicates topRightTree
        if ((topLeft.y + botRight.y) / 2 <= y)
        {
            return 1;
        }
        // Indicates botRightTree
        else
        {
            return 4;
        }
    }
}

__device__ bool inBoundary(Vector tl, Vector br, Vector p)
{
    return (p.x >= tl.x && p.x <= br.x && p.y <= tl.y && p.y >= br.y);
}

__device__ void updateBound(Vector &tl, Vector &br, int quadrant)
{
    if (quadrant == 1)
    {
        tl = {(tl.x + br.x) / 2, tl.y};
        br = {br.x, (tl.y + br.y) / 2};
    }
    else if (quadrant == 2)
    {
        tl = {tl.x, tl.y};
        br = {(tl.x + br.x) / 2, (tl.y + br.y) / 2};
    }
    else if (quadrant == 3)
    {
        tl = {tl.x, (tl.y + br.y) / 2};
        br = {(tl.x + br.x) / 2, br.y};
    }
    else
    {
        tl = {(tl.x + br.x) / 2, (tl.y + br.y) / 2};
        br = {br.x, br.y};
    }
}

__device__ void ConstructQuadTreeHelper(Node *node, int nodeIndex, Body body, Vector *topLeft, Vector *botRight, int *mutex, int nNodes, int nBodies, int leafLimit)
{

    Vector tl = *topLeft;
    Vector br = *botRight;

    while (nodeIndex < nNodes)
    {

        Node &curNode = node[nodeIndex];

        if (!inBoundary(tl, br, body.position))
        {
            break;
        }

        if (nodeIndex >= leafLimit)
        {
            // while (atomicCAS(&mutex[nodeIndex], 0, 1) != 0)
            //     ;
            if (curNode.centerMass.x != -1)
            {

                double M = curNode.totalMass + body.mass;
                double Rx = (curNode.totalMass * curNode.centerMass.x + body.mass * curNode.centerMass.x) / M;
                double Ry = (curNode.totalMass * curNode.centerMass.y + body.mass * curNode.centerMass.y) / M;
                curNode.totalMass = M;
                curNode.centerMass = {Rx, Ry};
            }
            else
            {
                curNode.totalMass = body.mass;
                curNode.centerMass = body.position;
            }
            // atomicExch(&mutex[nodeIndex], 0);
            break;
        }

        // If node x does not contain a body, put the new body here.
        if (curNode.isLeaf)
        {
            // while (atomicCAS(&mutex[nodeIndex], 0, 1) != 0)
            //     ;
            if (curNode.centerMass.x != -1)
            {

                int quadrant = getQuadrant(tl, br, curNode.centerMass.x, curNode.centerMass.y);
                Node &childNode = node[((nodeIndex * 4) + quadrant)];

                updateBound(tl, br, quadrant);
                childNode.centerMass = curNode.centerMass;
                childNode.totalMass = curNode.totalMass;

                curNode.centerMass = {-1, -1};
                curNode.totalMass = 0.0;
                curNode.isLeaf = false;
            }
            else
            {

                curNode.centerMass = body.position;
                curNode.totalMass = body.mass;
                // atomicExch(&mutex[nodeIndex], 0);
                break;
            }
            // atomicExch(&mutex[nodeIndex], 0);
        }

        int quadrant = getQuadrant(tl, br, body.position.x, body.position.y);
        updateBound(tl, br, quadrant);
        nodeIndex = (nodeIndex * 4) + quadrant;
    }
}

__global__ void ConstructQuadTreeKernel(Node *node, Body *bodies, Vector *topLeft, Vector *botRight, int *mutex, int nNodes, int nBodies, int leafLimit)
{

    int b = blockIdx.x * blockDim.x + threadIdx.x;

    if (b < nBodies)
    {
        Body body = bodies[b];
        ConstructQuadTreeHelper(node, 0, body, topLeft, botRight, mutex, nNodes, nBodies, leafLimit);
    }
}

// __device__ double getDistance(Vector pos1, Vector pos2)
// {

//     return sqrt(pow(pos1.x - pos2.x, 2) + pow(pos1.y - pos2.y, 2));
// }

// __device__ double getWidth(Node &root)
// {

//     return root.botRight.x - root.topLeft.x;
// }

// __device__ bool isCollide(Body &b1, Body &b2)
// {
//     return b1.radius + b2.radius > getDistance(b1.position, b2.position);
// }

// __device__ void ComputeForce(Node *node, int nodeIndex, Body *bodies, int bodyIndex, int nNodes, int nBodies)
// {
//     if (nodeIndex >= nNodes)
//     {
//         return;
//     }
//     Node &curNode = node[nodeIndex];
//     Body &bi = bodies[bodyIndex];
//     if (curNode.bi != -1)
//     {

//         Body &bj = bodies[curNode.bi];
//         if (isCollide(bi, bj))
//             return;

//         Vector rij = {bj.position.x - bi.position.x, bj.position.y - bi.position.y};
//         double inv_r3 = pow(rij.x * rij.x + rij.y * rij.y + E * E, -1.5);
//         double f = (GRAVITY * bj.mass) / inv_r3;
//         Vector force = {rij.x * f, rij.y * f};
//         bi.acceleration.x += (force.x / bi.mass);
//         bi.acceleration.y += (force.y / bi.mass);
//         return;
//     }

//     double sd = getWidth(curNode) / getDistance(bi.position, curNode.centerMass);
//     if (sd < THETA)
//     {
//         Vector rij = {curNode.centerMass.x - bi.position.x, curNode.centerMass.y - bi.position.y};
//         if (bi.radius * 2 > getDistance(bi.position, curNode.centerMass))
//             return;
//         double inv_r3 = pow(rij.x * rij.x + rij.y * rij.y + E * E, -1.5);
//         double f = (GRAVITY * curNode.totalMass) / inv_r3;
//         Vector force = {rij.x * f, rij.y * f};
//         bi.acceleration.x += (force.x / bi.mass);
//         bi.acceleration.y += (force.y / bi.mass);
//         return;
//     }

//     ComputeForce(node, (nodeIndex * 4) + 1, bodies, bodyIndex, nNodes, nBodies);
//     ComputeForce(node, (nodeIndex * 4) + 2, bodies, bodyIndex, nNodes, nBodies);
//     ComputeForce(node, (nodeIndex * 4) + 3, bodies, bodyIndex, nNodes, nBodies);
//     ComputeForce(node, (nodeIndex * 4) + 4, bodies, bodyIndex, nNodes, nBodies);
// }

// __global__ void ComputeForceKernel(Node *node, Body *bodies, int nNodes, int nBodies)
// {

//     int i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < nBodies)
//     {
//         Body &bi = bodies[i];
//         if (bi.isDynamic)
//         {

//             bi.velocity.x += bi.acceleration.x * DT / 2.0;
//             bi.velocity.y += bi.acceleration.y * DT / 2.0;

//             bi.position.x += bi.velocity.x * DT;
//             bi.position.y += bi.velocity.y * DT;

//             bi.acceleration = {0.0, 0.0};
//             ComputeForce(node, 0, bodies, i, nNodes, nBodies);
//             bi.velocity.x += bi.acceleration.x * DT / 2.0;
//             bi.velocity.y += bi.acceleration.y * DT / 2.0;
//         }
//     }
// }

#endif