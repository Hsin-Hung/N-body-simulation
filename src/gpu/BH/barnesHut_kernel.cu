#include "hip/hip_runtime.h"
#ifndef BARNES_HUT_KERNEL_
#define BARNES_HUT_KERNEL_

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>
#include "constants.h"
#include "barnesHutCuda.cuh"
#include "barnesHut_kernel.cuh"

/*
----------------------------------------------------------------------------------------
RESET KERNEL
----------------------------------------------------------------------------------------
*/
__global__ void ResetKernel(Node *node, Vector *topLeft, Vector *botRight, int *mutex, int nNodes, int nBodies)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;

    if (b < nNodes)
    {
        node[b].topLeft = {INFINITY, -INFINITY};
        node[b].botRight = {-INFINITY, INFINITY};
        node[b].centerMass = {-1, -1};
        node[b].totalMass = 0.0;
        node[b].isLeaf = true;
        node[b].start = -1;
        node[b].end = -1;
        mutex[b] = 0;
    }

    if (b == 0)
    {
        node[b].start = 0;
        node[b].end = nBodies - 1;
        *topLeft = {INFINITY, -INFINITY};
        *botRight = {-INFINITY, INFINITY};
    }
}

/*
----------------------------------------------------------------------------------------
COMPUTE BOUNDING BOX
----------------------------------------------------------------------------------------
*/
__global__ void ComputeBoundingBoxKernel(Node *node, Body *bodies, Vector *topLeft, Vector *botRight, int *mutex, int nBodies)
{

    __shared__ double topLeftX[BLOCK_SIZE];
    __shared__ double topLeftY[BLOCK_SIZE];
    __shared__ double botRightX[BLOCK_SIZE];
    __shared__ double botRightY[BLOCK_SIZE];

    int tx = threadIdx.x;
    int b = blockIdx.x * blockDim.x + tx;

    topLeftX[tx] = INFINITY;
    topLeftY[tx] = -INFINITY;
    botRightX[tx] = -INFINITY;
    botRightY[tx] = INFINITY;

    __syncthreads();

    if (b < nBodies)
    {
        Body body = bodies[b];
        topLeftX[tx] = body.position.x;
        topLeftY[tx] = body.position.y;
        botRightX[tx] = body.position.x;
        botRightY[tx] = body.position.y;
    }

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tx < s)
        {
            topLeftX[tx] = fminf(topLeftX[tx], topLeftX[tx + s]);
            topLeftY[tx] = fmaxf(topLeftY[tx], topLeftY[tx + s]);
            botRightX[tx] = fmaxf(botRightX[tx], botRightX[tx + s]);
            botRightY[tx] = fminf(botRightY[tx], botRightY[tx + s]);
        }
    }

    if (tx == 0)
    {
        while (atomicCAS(mutex, 0, 1) != 0)
            ;
        // topLeft->x = fminf(topLeft->x, topLeftX[0] - 1);
        // topLeft->y = fmaxf(topLeft->y, topLeftY[0] + 1);
        // botRight->x = fmaxf(botRight->x, botRightX[0] + 1);
        // botRight->y = fminf(botRight->y, botRightY[0] - 1);
        node[0].topLeft.x = fminf(node[0].topLeft.x, topLeftX[0] - 1);
        node[0].topLeft.y = fmaxf(node[0].topLeft.y, topLeftY[0] + 1);
        node[0].botRight.x = fmaxf(node[0].botRight.x, botRightX[0] + 1);
        node[0].botRight.y = fminf(node[0].botRight.y, botRightY[0] - 1);
        atomicExch(mutex, 0);
    }
}

/*
----------------------------------------------------------------------------------------
CONSTRUCT QUAD TREE
----------------------------------------------------------------------------------------
*/

__device__ int getQuadrant(Vector topLeft, Vector botRight, double x, double y)
{

    if ((topLeft.x + botRight.x) / 2 >= x)
    {
        // Indicates topLeftTree
        if ((topLeft.y + botRight.y) / 2 <= y)
        {
            return 2;
        }
        // Indicates botLeftTree
        else
        {
            return 3;
        }
    }
    else
    {
        // Indicates topRightTree
        if ((topLeft.y + botRight.y) / 2 <= y)
        {
            return 1;
        }
        // Indicates botRightTree
        else
        {
            return 4;
        }
    }
}

__device__ bool inBoundary(Vector tl, Vector br, Vector p)
{
    return (p.x >= tl.x && p.x <= br.x && p.y <= tl.y && p.y >= br.y);
}

__device__ void updateBound(Vector &tl, Vector &br, int quadrant)
{
    if (quadrant == 1)
    {
        tl = {(tl.x + br.x) / 2, tl.y};
        br = {br.x, (tl.y + br.y) / 2};
    }
    else if (quadrant == 2)
    {
        tl = {tl.x, tl.y};
        br = {(tl.x + br.x) / 2, (tl.y + br.y) / 2};
    }
    else if (quadrant == 3)
    {
        tl = {tl.x, (tl.y + br.y) / 2};
        br = {(tl.x + br.x) / 2, br.y};
    }
    else
    {
        tl = {(tl.x + br.x) / 2, (tl.y + br.y) / 2};
        br = {br.x, br.y};
    }
}

__device__ void ConstructQuadTreeHelper(Node *node, int nodeIndex, Body body, Vector *topLeft, Vector *botRight, int *mutex, int nNodes, int nBodies, int leafLimit)
{

    Vector tl = *topLeft;
    Vector br = *botRight;

    while (nodeIndex < nNodes)
    {

        Node &curNode = node[nodeIndex];

        if (!inBoundary(tl, br, body.position))
        {
            break;
        }

        if (nodeIndex >= leafLimit)
        {

            if (curNode.centerMass.x != -1)
            {

                double M = curNode.totalMass + body.mass;
                double Rx = (curNode.totalMass * curNode.centerMass.x + body.mass * curNode.centerMass.x) / M;
                double Ry = (curNode.totalMass * curNode.centerMass.y + body.mass * curNode.centerMass.y) / M;
                curNode.totalMass = M;
                curNode.centerMass = {Rx, Ry};
            }
            else
            {
                curNode.totalMass = body.mass;
                curNode.centerMass = body.position;
            }
            break;
        }

        // If node x does not contain a body, put the new body here.
        if (curNode.isLeaf)
        {
            if (curNode.centerMass.x != -1)
            {

                int quadrant = getQuadrant(tl, br, curNode.centerMass.x, curNode.centerMass.y);
                Node &childNode = node[(nodeIndex * 4) + quadrant];
                childNode.centerMass = curNode.centerMass;
                childNode.totalMass = curNode.totalMass;

                curNode.centerMass = {-1, -1};
                curNode.totalMass = 0.0;
                curNode.isLeaf = false;
            }
            else
            {

                curNode.centerMass = body.position;
                curNode.totalMass = body.mass;
                break;
            }
        }

        int quadrant = getQuadrant(tl, br, body.position.x, body.position.y);
        updateBound(tl, br, quadrant);
        nodeIndex = (nodeIndex * 4) + quadrant;
    }
}

__global__ void ConstructQuadTreeKernel(Node *node, Body *bodies, Vector *topLeft, Vector *botRight, int *mutex, int nNodes, int nBodies, int leafLimit)
{

    int b = blockIdx.x * blockDim.x + threadIdx.x;

    if (b < nBodies)
    {
        Body body = bodies[b];
        ConstructQuadTreeHelper(node, 0, body, topLeft, botRight, mutex, nNodes, nBodies, leafLimit);
    }
}

__device__ void CountBodies(Body *bodies, Vector topLeft, Vector botRight, int *count, int start, int end, int nBodies)
{
    if (threadIdx.x < 4)
        count[threadIdx.x] = 0;
    __syncthreads();
    if (threadIdx.x == 0)
    {

        for (int i = start; i <= end; ++i)
        {
            Body &body = bodies[i];
            int quadrant = getQuadrant(topLeft, botRight, body.position.x, body.position.y);
            ++count[quadrant - 1];
        }
    }

    __syncthreads();
}

__device__ void GroupBodies(Body *bodies, Body *buffer, Vector topLeft, Vector botRight, int *count, int start, int end, int nBodies)
{
    int q1 = start, q2 = start + count[0], q3 = start + count[0] + count[1], q4 = start + count[0] + count[1] + count[2];
    if (threadIdx.x == 0)
    {

        for (int i = start; i <= end; ++i)
        {
            Body &body = bodies[i];
            int quadrant = getQuadrant(topLeft, botRight, body.position.x, body.position.y);
            if (quadrant == 1)
            {

                buffer[q1++] = body;
            }
            else if (quadrant == 2)
            {

                buffer[q2++] = body;
            }
            else if (quadrant == 3)
            {

                buffer[q3++] = body;
            }
            else
            {

                buffer[q4++] = body;
            }
        }
    }

    __syncthreads();
}

__global__ void ConstructQuadTreeDPKernel(Node *node, Body *bodies, Body *buffer, int nodeIndex, int nNodes, int nBodies, int leafLimit)
{
    __shared__ int count[4];
    int tx = threadIdx.x;
    nodeIndex += blockIdx.x;

    if (nodeIndex >= nNodes)
        return;

    Node &curNode = node[nodeIndex];
    int start = curNode.start, end = curNode.end;
    // if (tx == 0)
    // {
    //     printf("node: %d, start-end: %d -> %d , topleft x: %f y: %f -> botright x: %f y: %f\n", nodeIndex, start, end,
    //            curNode.topLeft.x, curNode.topLeft.y, curNode.botRight.x, curNode.botRight.y);
    // }

    Vector topLeft = curNode.topLeft, botRight = curNode.botRight;

    if (start == -1 && end == -1)
        return;

    double M = curNode.totalMass;
    double Rx = curNode.totalMass * curNode.centerMass.x;
    double Ry = curNode.totalMass * curNode.centerMass.y;
    for (int i = start; i <= end; ++i)
    {
        Body &body = bodies[i];
        M += body.mass;
        Rx += body.mass * body.position.x;
        Ry += body.mass * body.position.y;
    }
    Rx /= M;
    Ry /= M;
    curNode.totalMass = M;
    curNode.centerMass = {Rx, Ry};

    // if (tx == 0)
    //     printf("node: %d, total mass: %f, center mass: %f, %f\n", nodeIndex, M, Rx, Ry);

    if (nodeIndex >= leafLimit || start == end)
    {
        buffer[start] = bodies[start];
        return;
    }

    // if (threadIdx.x == 0)
    // {
    //     for (int i = start; i <= end; ++i)
    //     {
    //         printf("before node: %d, body id: %d \n", nodeIndex, bodies[i].id);
    //     }
    // }
    CountBodies(bodies, topLeft, botRight, count, start, end, nBodies);
    GroupBodies(bodies, buffer, topLeft, botRight, count, start, end, nBodies);
    // if (threadIdx.x == 0)
    // {
    //     for (int i = start; i <= end; ++i)
    //     {
    //         printf("after node: %d, body id: %d \n", nodeIndex, buffer[i].id);
    //     }
    // }
    Node &topLNode = node[(nodeIndex * 4) + 1],
         &topRNode = node[(nodeIndex * 4) + 2], &botLNode = node[(nodeIndex * 4) + 3], &botRNode = node[(nodeIndex * 4) + 4];

    if (tx == 0)
    {
        // printf("node: %d, count: %d %d %d %d\n", nodeIndex, count[0], count[1], count[2], count[3]);
        topLNode.topLeft = topLeft;
        topLNode.botRight = botRight;
        topRNode.topLeft = topLeft;
        topRNode.botRight = botRight;
        botLNode.topLeft = topLeft;
        botLNode.botRight = botRight;
        botRNode.topLeft = topLeft;
        botRNode.botRight = botRight;

        updateBound(topLNode.topLeft, topLNode.botRight, 1);
        updateBound(topRNode.topLeft, topRNode.botRight, 2);
        updateBound(botLNode.topLeft, botLNode.botRight, 3);
        updateBound(botRNode.topLeft, botRNode.botRight, 4);

        curNode.isLeaf = false;

        if (count[0] > 0)
        {
            topLNode.start = start;
            topLNode.end = start + count[0] - 1;
        }

        if (count[1] > 0)
        {
            topRNode.start = start + count[0];
            topRNode.end = start + count[0] + count[1] - 1;
        }

        if (count[2] > 0)
        {
            botLNode.start = start + count[0] + count[1];
            botLNode.end = start + count[0] + count[1] + count[2] - 1;
        }

        if (count[3] > 0)
        {
            botRNode.start = start + count[0] + count[1] + count[2];
            botRNode.end = end;
        }
        ConstructQuadTreeDPKernel<<<4, BLOCK_SIZE>>>(node, buffer, bodies, nodeIndex * 4 + 1, nNodes, nBodies, leafLimit);
    }
}

/*
----------------------------------------------------------------------------------------
COMPUTE CENTER MASS
----------------------------------------------------------------------------------------
*/

__global__ void ComputeCenterMass(Node *node, int nNodes, int start, int end)
{
    int nodeIndex = blockIdx.x * blockDim.x + threadIdx.x;
    nodeIndex += start;
    if (nodeIndex >= start && nodeIndex < end)
    {

        Node &curNode = node[nodeIndex];

        if (!curNode.isLeaf)
        {

            Node &topLNode = node[(nodeIndex * 4) + 2], &topRNode = node[(nodeIndex * 4) + 1], &botLNode = node[(nodeIndex * 4) + 3], &botRNode = node[(nodeIndex * 4) + 4];
            double totalChildMass = topLNode.totalMass + topRNode.totalMass + botLNode.totalMass + botRNode.totalMass;
            double totalCenterMassX = 0.0, totalCenterMassY = 0.0;

            totalCenterMassX += topLNode.centerMass.x * topLNode.totalMass;
            totalCenterMassY += topLNode.centerMass.y * topLNode.totalMass;

            totalCenterMassX += topRNode.centerMass.x * topRNode.totalMass;
            totalCenterMassY += topRNode.centerMass.y * topRNode.totalMass;

            totalCenterMassX += botLNode.centerMass.x * botLNode.totalMass;
            totalCenterMassY += botLNode.centerMass.y * botLNode.totalMass;

            totalCenterMassX += botRNode.centerMass.x * botRNode.totalMass;
            totalCenterMassY += botRNode.centerMass.y * botRNode.totalMass;

            curNode.totalMass = totalChildMass;
            curNode.centerMass = {totalCenterMassX / totalChildMass, totalCenterMassY / totalChildMass};
        }
    }
}

// /*
// ----------------------------------------------------------------------------------------
// COMPUTE FORCE
// ----------------------------------------------------------------------------------------
// */

__device__ double getDistance(Vector pos1, Vector pos2)
{

    return sqrt(pow(pos1.x - pos2.x, 2) + pow(pos1.y - pos2.y, 2));
}

__device__ bool isCollide(Body &b1, Body &b2)
{
    return b1.radius + b2.radius > getDistance(b1.position, b2.position);
}
__device__ void ComputeForceRecursive(Node *node, Body *bodies, int nodeIndex, int bodyIndex, int nNodes, int nBodies, int leafLimit, double width)
{

    if (nodeIndex >= nNodes)
    {
        return;
    }
    Node &curNode = node[nodeIndex];
    Body &bi = bodies[bodyIndex];
    if (curNode.isLeaf)
    {

        if (curNode.centerMass.x != -1)
        {
            if (bi.radius * 2 > getDistance(bi.position, curNode.centerMass))
                return;

            Vector rij = {curNode.centerMass.x - bi.position.x, curNode.centerMass.y - bi.position.y};
            double inv_r3 = pow(rij.x * rij.x + rij.y * rij.y + E * E, -1.5);
            double f = (GRAVITY * curNode.totalMass) / inv_r3;
            Vector force = {rij.x * f, rij.y * f};
            bi.acceleration.x += (force.x / bi.mass);
            bi.acceleration.y += (force.y / bi.mass);
        }
        return;
    }

    double sd = width / getDistance(bi.position, curNode.centerMass);
    if (sd < THETA)
    {
        Vector rij = {curNode.centerMass.x - bi.position.x, curNode.centerMass.y - bi.position.y};
        if (bi.radius * 2 > getDistance(bi.position, curNode.centerMass))
            return;
        double inv_r3 = pow(rij.x * rij.x + rij.y * rij.y + E * E, -1.5);
        double f = (GRAVITY * curNode.totalMass) / inv_r3;
        Vector force = {rij.x * f, rij.y * f};
        bi.acceleration.x += (force.x / bi.mass);
        bi.acceleration.y += (force.y / bi.mass);
        return;
    }

    ComputeForceRecursive(node, bodies, (nodeIndex * 4) + 1, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForceRecursive(node, bodies, (nodeIndex * 4) + 2, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForceRecursive(node, bodies, (nodeIndex * 4) + 3, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
    ComputeForceRecursive(node, bodies, (nodeIndex * 4) + 4, bodyIndex, nNodes, nBodies, leafLimit, width / 2);
}
__device__ void ComputeForce(Node *node, Body *bodies, int bodyIndex, int nNodes, int nBodies, int leafLimit, double width)
{
    Body &bi = bodies[bodyIndex];
    int q_size = 1024;
    __shared__ int queue[1024];
    int front = 0, insert = 0;
    int size;
    queue[insert++] = 0;

    while (front != insert)
    {
        size = insert - front;

        for (int i = 0; i < size; ++i)
        {

            int nodeIndex = queue[front++];
            front %= q_size;
            Node &curNode = node[nodeIndex];

            if (curNode.isLeaf)
            {
                if (curNode.centerMass.x != -1)
                {
                    if (bi.radius * 2 > getDistance(bi.position, curNode.centerMass))
                        continue;

                    Vector rij = {curNode.centerMass.x - bi.position.x, curNode.centerMass.y - bi.position.y};
                    double inv_r3 = pow(rij.x * rij.x + rij.y * rij.y + E * E, -1.5);
                    double f = (GRAVITY * curNode.totalMass) / inv_r3;
                    Vector force = {rij.x * f, rij.y * f};
                    bi.acceleration.x += (force.x / bi.mass);
                    bi.acceleration.y += (force.y / bi.mass);
                }
                continue;
            }

            double sd = width / getDistance(bi.position, curNode.centerMass);
            if (sd < THETA)
            {
                Vector rij = {curNode.centerMass.x - bi.position.x, curNode.centerMass.y - bi.position.y};
                if (bi.radius * 2 > getDistance(bi.position, curNode.centerMass))
                    continue;
                double inv_r3 = pow(rij.x * rij.x + rij.y * rij.y + E * E, -1.5);
                double f = (GRAVITY * curNode.totalMass) / inv_r3;
                Vector force = {rij.x * f, rij.y * f};
                bi.acceleration.x += (force.x / bi.mass);
                bi.acceleration.y += (force.y / bi.mass);
                continue;
            }

            if (node[(nodeIndex * 4) + 1].totalMass > 0)
                queue[insert++] = (nodeIndex * 4) + 1;
            if (node[(nodeIndex * 4) + 2].totalMass > 0)
                queue[insert++] = (nodeIndex * 4) + 2;
            if (node[(nodeIndex * 4) + 3].totalMass > 0)
                queue[insert++] = (nodeIndex * 4) + 3;
            if (node[(nodeIndex * 4) + 4].totalMass > 0)
                queue[insert++] = (nodeIndex * 4) + 4;
            insert %= q_size;
        }

        width /= 2.0;
    }
}

__global__ void ComputeForceKernel(Node *node, Body *bodies, int nNodes, int nBodies, int leafLimit)
{

    int i = blockIdx.x;
    double width = node[0].botRight.x - node[0].topLeft.x;

    if (i < nBodies)
    {
        Body &bi = bodies[i];
        if (bi.isDynamic)
        {
            if (threadIdx.x == 0)
            {
                bi.velocity.x += bi.acceleration.x * DT / 2.0;
                bi.velocity.y += bi.acceleration.y * DT / 2.0;
                // printf("velocity: x: %f  y: %f\n", bi.velocity.x, bi.velocity.y);
                bi.position.x += bi.velocity.x * DT;
                bi.position.y += bi.velocity.y * DT;

                bi.acceleration = {0.0, 0.0};

                ComputeForceRecursive(node, bodies, 0, i, nNodes, nBodies, leafLimit, width);
                bi.velocity.x += bi.acceleration.x * DT / 2.0;
                bi.velocity.y += bi.acceleration.y * DT / 2.0;
            }
        }
    }
}

#endif